
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512
#define RADIUS 3


__global__ void add(int *in,int *out,int size) { 
  __shared__ int temp[THREADS_PER_BLOCK + (2*RADIUS)];
  int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int localIdx = threadIdx.x + RADIUS;
  int localSum = 0 ;
  temp[localIdx] = in[globalIdx];
  if(threadIdx.x < RADIUS) {
    if((globalIdx - RADIUS) >= 1) {
      temp[localIdx - RADIUS] = in[globalIdx - RADIUS];
    }else {
      temp[localIdx - RADIUS] = 0;
    }
    if((globalIdx + THREADS_PER_BLOCK) < size) {
    temp[localIdx + THREADS_PER_BLOCK] = in[globalIdx + THREADS_PER_BLOCK];
    }else {
      temp[localIdx + THREADS_PER_BLOCK] = 0;
    }
  }
  __syncthreads();
  for(int i = -RADIUS; i  <= RADIUS; i++) {
    localSum = localSum + temp[threadIdx.x + RADIUS  + i];
  }
  out[globalIdx] = localSum;
  __syncthreads();
} 

int main(void) {
  int *a,*b; 
  int *d_a,*d_b;
  int size = N * sizeof(int);
  hipMalloc((void **)&d_a,size);
  hipMalloc((void **)&d_b,size); 
  a = (int *)malloc(size);
  b = (int *)malloc(size); 

  for(int i = 0; i < N;i++) {
    a[i] = 1;
  }

  hipMemcpy(d_a,a,size,hipMemcpyHostToDevice); 
  add<<<(N + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,N);
  hipMemcpy(b,d_b,size,hipMemcpyDeviceToHost); 
  printf("Hello world %d\n",b[120]);
  free(a);
  free(b);
  hipFree(d_a);
  hipFree(d_b); 
  return 0;
}
