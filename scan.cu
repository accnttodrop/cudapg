
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 1024
#define SIZE 5000000
#define LL long long int

__global__ void scan(LL* data,LL* result, unsigned int size) {
  unsigned int  globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if(globalIdx < size) { 
    result[globalIdx] = data[globalIdx]; 
  }else {

  }
}

LL* AllocArr(unsigned int size); 
int main(void) {
  LL *iptr,*optr = NULL;
  LL *di_ptr,*do_ptr = NULL;
  unsigned int  size = SIZE;
  unsigned int  memSize = SIZE * sizeof(LL);
  int  result = 0; 
  iptr = AllocArr(size);
  optr = (LL *)malloc(memSize); 
  result = hipMalloc((void **)&di_ptr,memSize);
  result = hipMalloc((void **)&do_ptr,memSize); 
  result = hipMemcpy(di_ptr,iptr,memSize,hipMemcpyHostToDevice); 
  scan<<<(SIZE + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(di_ptr,do_ptr,size);
  hipMemcpy(optr,do_ptr,memSize,hipMemcpyDeviceToHost); 
  printf("Non computed %lld \nComputed %lld\n",iptr[20],optr[20]);
  free(iptr); 
  free(optr); 
  hipFree(di_ptr);
  hipFree(do_ptr); 
  return 1;
}

LL* AllocArr(unsigned int size) {
  LL * ptr = NULL;
  ptr = (LL *)malloc(sizeof(LL)*size);
  unsigned int i = 0;
  for(i = 0; i < size;i++) {
    ptr[i] = 1;
  }
  return ptr;
}


