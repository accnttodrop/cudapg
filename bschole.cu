
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a,int *b,int *c,int n) { 
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < n) {
    c[index] = a[index] + b[index]; 
  }
}

int main(void) {
  int *a,*b,*c; 
  int *d_a,*d_b,*d_c;
  int size = N * sizeof(int);
  hipMalloc((void **)&d_a,size);
  hipMalloc((void **)&d_b,size); 
  hipMalloc((void **)&d_c,size); 
  a = (int *)malloc(size);
  b = (int *)malloc(size); 
  c = (int *)malloc(size); 
  for(int i = 0; i < N;i++) {
    a[i] = i+1;
    b[i] = i+1; 
  }

  hipMemcpy(d_a,a,size,hipMemcpyHostToDevice); 
  hipMemcpy(d_b,b,size,hipMemcpyHostToDevice); 
  add<<<(N + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c,N);
  hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost); 
  printf("Hello world %d\n",c[100]);
  free(a);
  free(b);
  free(c);
  hipFree(d_a);
  hipFree(d_b); 
  hipFree(d_c); 
  return 0;
}
