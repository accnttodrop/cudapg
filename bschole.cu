
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 256
#define THREADS_PER_BLOCK 256
//Test
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

__global__ void add(double *price,int n) { 
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ double returns[256]; 
  //Calculate Log Returns
  double logRt = 0.0;
  if(index == 0) {
    returns[0] = 0.0;
  }else if(index < n) {
    logRt = log(price[index]) - log(price[index-1]);
    returns[index] = logRt; 
  }
  __syncthreads();

  
  //find average of returns
  int idx = 2; 
  int back = 1;
  while(idx <= (n+1)) { 
  if((index+1) % idx == 0) { 
    returns[index] = returns[index] + returns[index-back]; 
  }
  idx = idx * 2;
  back = back * 2;
    __syncthreads(); 
  }
  __syncthreads();

  float ravg = returns[n-1]/n; 
  float rdiffSq = (logRt - ravg) * (logRt - ravg); 
  __syncthreads(); 
  returns[index] = rdiffSq; 
  __syncthreads(); 
  idx = 2; 
  back = 1;
  while(idx <= (n + 1)) { 
  if((index+1) % idx == 0) { 
      returns[index] = returns[index] + returns[index-back]; 
    }
  idx = idx * 2;
  back = back * 2;
    __syncthreads(); 
  }
  __syncthreads();
  if(index == 0) {
    float vol  = returns[n-1]/(n-2);
    float sd = sqrt(vol); 
    printf("SD  %f Volatility   %f\n",sd,vol); 
  }
}
  

int main(void) {
  double *price; 
  double *d_price;
  int size = N * sizeof(double);
  hipMalloc((void **)&d_price,size);

  price = (double *)malloc(size);

  for(int i = 0; i < N;i++) {
    price[i] = i+1;
  }

  hipMemcpy(d_price,price,size,hipMemcpyHostToDevice); 
  add<<<(N + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_price,N);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() ); 
  free(price);
  hipFree(d_price);
  return 0;
}
