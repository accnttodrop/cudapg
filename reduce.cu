
#include <hip/hip_runtime.h>
#include "stdio.h"

void generateData(int a,int **b); 

int main() { 
  int count = 100; 
  int* elements; 
  generateData(count,&elements); 
  int sum = 0;
  for(int i = 0;i < count;i++) {
    sum = sum + elements[i];
  }
  printf("Sum %d\n",sum); 
  free(elements);
  return 0;
} 


void generateData(int totalCount,int **ptr) {
  *ptr = NULL; 
  *ptr = (int *) malloc(totalCount * sizeof(int));
  for(int i = 0;i < totalCount; i++) {
    *((*ptr) + i) = i;
  }
  printf("Data generated\n");
}
    
