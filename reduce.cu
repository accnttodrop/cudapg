
#include <hip/hip_runtime.h>
#include "stdio.h"

void generateData(int a,float **b); 

typedef struct {
  int  id;
  int  binId1;
  int  binId2;
  float *price;
    } Security; 

void generateData(int totalCount,float **ptr) {
  *ptr = NULL; 
  *ptr = (float *) malloc(totalCount * sizeof(float));
  for(int i = 0;i < totalCount; i++) {
    *((*ptr) + i) = 20.0+((double)rand()/(double)(RAND_MAX-1))*100;
  }
}

void generateSecurityData(int perBin,int bins,Security **securities) {
  int total = 1;
  *securities = NULL;
  *securities = (Security *)malloc(perBin*bins*sizeof(Security));
  for(int i =0;i < bins; i++) {
    for(int q = 0; q < perBin; q++) {
      Security sec;
      sec.id = total+1;
      sec.binId1 = (i+1); 
      generateData(2048,&sec.price);
      *((*securities + total)) = sec;
    }
  }
}   

int main() { 
  Security *securities;
  generateSecurityData(256,32,&securities);
  return 0;
} 
